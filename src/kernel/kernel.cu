#include "hip/hip_runtime.h"
#include "kernel.cuh"

#include <assert.h>
#include <iostream>
#include <stdio.h>
#include "gpuErrchk.cuh"

#include "..\matrix_loader\matrix.hpp"

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	return threadId;
}

template<class matrix>
__device__ matrix sumMatrix(const matrix * const d_inputMatrix1, const matrix * const d_inputMatrix2, int index)
{
	return d_inputMatrix1[index] + d_inputMatrix2[index];
}

template<class matrix, class error>
__device__ void injectError(matrix * const d_inputMatrix, const error * const d_errorMap, int index)
{
	d_inputMatrix[index] += d_errorMap[index];
}

template<class matrix, class error>
__global__ void matrixOperation(const matrix  * const d_inputMatrix1, const matrix  * const d_inputMatrix2, matrix * const d_outputMatrix, const error * const d_errorMap)
{
	int index = getGlobalIdx_2D_2D();
	d_outputMatrix[index] = sumMatrix(d_inputMatrix1, d_inputMatrix2, index);
	injectError(d_outputMatrix, d_errorMap, index);
}

template <class error>
void fillErrorMap(error * const errorMap, const int numRows, const int numCols)
{
	for (int i = 0; i < numRows*numCols; i++)
	{
		errorMap[i] = 10;
	}
}


void startKernel()
{
	// C++11 not supported by CUDA (can't use smart pointers)
	Matrix * h_in1 = new Matrix("matrixes/bcsstk03.mtx");
	Matrix * h_in2 = new Matrix("matrixes/bcsstk03.mtx");

	// Make sure arrays have same dimensions
	assert(h_in1->getColumns() == h_in2->getColumns());
	assert(h_in1->getRows() == h_in2->getRows());

	const int ARRAY_SIZE = h_in1->getColumns()*h_in1->getRows();
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);


	// CUDA SM 1.1 doesn't support double, need to convert to floats
	float * h_in1_float = new float[ARRAY_SIZE];
	for (int i = 0; i < 112 * 112; i++)
	{
		h_in1_float[i] = static_cast<float>(h_in1->getMatrix()[i]);
	}

	float * h_in2_float = new float[ARRAY_SIZE];
	for (int i = 0; i < 112 * 112; i++)
	{
		h_in2_float[i] = static_cast<float>(h_in1->getMatrix()[i]);
	}



	float * h_out = new float[ARRAY_SIZE];

	float * h_error = new float[ARRAY_SIZE];
	fillErrorMap(h_error, h_in1->getRows(), h_in1->getColumns());


	


	for (int i = 0; i < 10; i++)
	{	std::cout << "h1  -> " << h_in1_float[i] << "; ";
		std::cout << "h2  -> " << h_in2_float[i] << "; ";
		std::cout << "err -> " << h_error[i] << ";";

		std::cout << "suma ->" << h_in1_float[i] + h_in1_float[i] + h_error[i] << std::endl;
	}

	float * d_in1;
	float * d_in2;
	float * d_out;
	float * d_error;
	

	gpuErrchk(hipMalloc((void**)&d_in1, ARRAY_BYTES));
	gpuErrchk(hipMalloc((void**)&d_in2, ARRAY_BYTES));
	gpuErrchk(hipMalloc((void**)&d_out, ARRAY_BYTES));
	gpuErrchk(hipMalloc((void**)&d_error, ARRAY_BYTES));


	//gpuErrchk(hipMemcpy(d_in1, h_in1->getMatrix(), ARRAY_BYTES, hipMemcpyHostToDevice));
	//gpuErrchk(hipMemcpy(d_in2, h_in2->getMatrix(), ARRAY_BYTES, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_in1, h_in1_float, ARRAY_BYTES, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_in2, h_in2_float, ARRAY_BYTES, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_error, h_error, ARRAY_BYTES, hipMemcpyHostToDevice));

	const dim3 gridSize(112, 112, 1);  
	const dim3 blockSize(1, 1, 1);  
	matrixOperation <<< gridSize, blockSize >>>(d_in1, d_in2, d_out, d_error);
	
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost));
	
	std::cout << "out:" << std::endl;
	for (int i = 0; i < 112; i++)
		std::cout << "h_out -> " << h_out[i] << std::endl;

	hipFree(d_in1);
	hipFree(d_in2);
	hipFree(d_out);
	hipFree(d_error);


}

