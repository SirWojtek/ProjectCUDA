#include "hip/hip_runtime.h"

#include "kernel.cuh"
#include "gpuErrchk.cuh"

#include <assert.h>
#include <iostream>
#include <stdio.h>

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	return threadId;
}

__device__ float sumMatrix(const float * const d_inputMatrix1, const float * const d_inputMatrix2, int index)
{
	return d_inputMatrix1[index] + d_inputMatrix2[index];
}

__device__ void injectError(float * const d_inputMatrix, const float * const d_errorMap, int index)
{
	d_inputMatrix[index] += d_errorMap[index];
}

__global__ void matrixOperation(const float  * const d_inputMatrix1,
	const float  * const d_inputMatrix2, float * const d_outputMatrix)
{
	int index = getGlobalIdx_2D_2D();
	d_outputMatrix[index] = sumMatrix(d_inputMatrix1, d_inputMatrix2, index);
}

__global__ void matrixOperation(const float  * const d_inputMatrix1,
	const float  * const d_inputMatrix2, float * const d_outputMatrix, float * const d_errorMap)
{
	int index = getGlobalIdx_2D_2D();
	d_outputMatrix[index] = sumMatrix(d_inputMatrix1, d_inputMatrix2, index);
	injectError(d_outputMatrix, d_errorMap, getGlobalIdx_2D_2D());
}

void fillErrorMap(float * const errorMap, const int numRows, const int numCols)
{
	for (int i = 0; i < numRows*numCols; i++)
	{
		errorMap[i] = 10;
	}
}

void runKernel(dim3 gridSize, dim3 blockSize, float* in1, float* in2, float* out)
{
	//std::cout << gridSize.x << gridSize.y << gridSize.z << "\n";
	//std::cout << blockSize.x << blockSize.y << blockSize.z << "\n";
	hipEvent_t start, stop; // Mam pewne obawy przed wyrzucaniem tego do osobnych funkcji, �eby nie zajmowa�o niepotrzebnie czasu systemowego
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	matrixOperation <<< gridSize, blockSize >>>(in1, in2, out);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	std::cout << "Time for the Kernel: " << time << std::endl;
}

void runKernelWithError(dim3 gridSize, dim3 blockSize, float* in1, float* in2,
	float* out, float* error)
{
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	matrixOperation <<< gridSize, blockSize >>>(in1, in2, out, error);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);
	std::cout << "Time for the ErrorKernel: " << time << std::endl;
}
