#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "..\kernelCommon\gpuErrchk.cuh"
#include "..\..\matrix_loader\matrix.hpp"
#include <assert.h>
#include <iostream>
#include <stdio.h>


// broken kernel
__device__ int brokenBlock = 0;
__device__ int brokenThread = 15;

__device__ int getErrorIdx_1D_1D()
{
	return brokenBlock *blockDim.x + brokenThread;
}

__device__ int getGlobalIdx_1D_1D()
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__device__ int getGlobalIdx_2D_2D()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	
	return threadId;
}

__device__ CellInfo sumMatrix(const CellInfo * const d_inputMatrix1, const CellInfo * const d_inputMatrix2, int index)
{
	CellInfo result = {
		d_inputMatrix1[index].value + d_inputMatrix2[index].value,
		d_inputMatrix1[index].row,
		d_inputMatrix1[index].column
						};
	return result;
}

__device__ float sumMatrix(const float * const d_inputMatrix1, const float * const d_inputMatrix2, int index)
{
	float result = d_inputMatrix1[index] + d_inputMatrix2[index];
	return result;
}

__device__ bool isError(int index)
{
	if (index == getErrorIdx_1D_1D())
		return true;
	else
		return false;
}

__device__ void injectError(CellInfo &inputCell)
{
	inputCell.value += 99999999;
}

__device__ void injectError(float *inputCell)
{
	*inputCell += 99999999;
}

__global__ void kernel(const CellInfo  * const d_inputMatrix1,
	const CellInfo  * const d_inputMatrix2, CellInfo * const d_outputMatrix)
{
	int index = getGlobalIdx_1D_1D();
	d_outputMatrix[index] = sumMatrix(d_inputMatrix1, d_inputMatrix2, index);
}

__global__ void kernel(const float  * const d_inputMatrix1,
	const float  * const d_inputMatrix2, float * const d_outputMatrix)
{
	int index = getGlobalIdx_1D_1D();
	d_outputMatrix[index] = sumMatrix(d_inputMatrix1, d_inputMatrix2, index);
}

__global__ void kernelWithCounter(const float  * const d_inputMatrix1,
	const float  * const d_inputMatrix2, float * const d_outputMatrix, int *current_thread_count)
{
	atomicAdd(current_thread_count, 1);
	int index = getGlobalIdx_1D_1D();
	d_outputMatrix[index] = sumMatrix(d_inputMatrix1, d_inputMatrix2, index);
}

void runKernel(dim3 gridSize, dim3 blockSize, const float  * const d_inputMatrix1,
	const float  * const d_inputMatrix2, float * const d_outputMatrix)
{
	kernel << < gridSize, blockSize >> > (d_inputMatrix1, d_inputMatrix2, d_outputMatrix);
}

__global__ void kernelPlusError(const CellInfo  * const d_inputMatrix1,
	const CellInfo  * const d_inputMatrix2, CellInfo * const d_outputMatrix)
{
	int index = getGlobalIdx_1D_1D();
	d_outputMatrix[index] = sumMatrix(d_inputMatrix1, d_inputMatrix2, index);
	if (isError(index))
		injectError(d_outputMatrix[index]);
}

__global__ void kernelPlusError(const float  * const d_inputMatrix1,
	const float  * const d_inputMatrix2, float * const d_outputMatrix)
{
	int index = getGlobalIdx_1D_1D();
	d_outputMatrix[index] = sumMatrix(d_inputMatrix1, d_inputMatrix2, index);
	if (isError(index))
		injectError(&d_outputMatrix[index]);
}

void runKernelPlusError(dim3 gridSize,
	const float * hostInputMatrix1, const float * hostInputMatrix2,
	float* hostOutputMatrix)
{
	int arrayBytes = gridSize.x * sizeof(float);
	float * inputMatrix1;
	float * inputMatrix2;
	float * outputMatrix;
	dim3 blockSize(1, 1, 1);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float timer;

	gpuErrchk(hipMalloc((void**)&inputMatrix1, arrayBytes));
	gpuErrchk(hipMalloc((void**)&inputMatrix2, arrayBytes));
	gpuErrchk(hipMalloc((void**)&outputMatrix, arrayBytes));

	hipEventRecord(start, 0);

	gpuErrchk(hipMemcpy(inputMatrix1, hostInputMatrix1, arrayBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(inputMatrix2, hostInputMatrix2, arrayBytes, hipMemcpyHostToDevice));

	kernelPlusError <<< gridSize, blockSize >>> (inputMatrix1, inputMatrix2, outputMatrix);

	gpuErrchk(hipMemcpy(hostOutputMatrix, outputMatrix, arrayBytes, hipMemcpyDeviceToHost));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timer, start, stop);

	std::cout << "Error calculation time [ms]: " << timer << std::endl;
	std::cout << "No redundant calculation in this scope." << std::endl;

	hipFree(inputMatrix1);
	hipFree(inputMatrix2);
	hipFree(outputMatrix);
}

void runCommandCenter(dim3 gridSize, dim3 redundantGridSize,
	const float * hostInputMatrix1, const float * hostInputMatrix2,
	float* hostOutputMatrix, float* hostRedundantMatrix)
{
	if (redundantGridSize.x == 0)
	{
		runKernelPlusError(gridSize, hostInputMatrix1, 
			hostInputMatrix2, hostOutputMatrix);
		return;
	}
	int arrayBytes = gridSize.x * sizeof(float);
	int arrayBytesRedundant = redundantGridSize.x * sizeof(float);
	dim3 blockSize(1, 1, 1);
	hipStream_t stream[2];

	// input (device)
	float * inputMatrix1;
	float * inputMatrix2;
	float * redundantMatrix1;
	float * redundantMatrix2;
	// output (device)
	float * outputMatrix1;
	float * outputMatrix2;


	gpuErrchk(hipMalloc((void**)&inputMatrix1, arrayBytes));
	gpuErrchk(hipMalloc((void**)&inputMatrix2, arrayBytes));
	gpuErrchk(hipMalloc((void**)&outputMatrix1, arrayBytes));
	gpuErrchk(hipMalloc((void**)&outputMatrix2, arrayBytesRedundant));
	gpuErrchk(hipMalloc((void**)&redundantMatrix1, arrayBytesRedundant));
	gpuErrchk(hipMalloc((void**)&redundantMatrix2, arrayBytesRedundant));

	hipEvent_t start[2], stop[2];
	float timer[2];

	hipStreamCreate(&stream[0]);
	hipStreamCreate(&stream[1]);

	hipEventCreate(&start[0]);
  	hipEventRecord(start[0], stream[0]);

	gpuErrchk(hipMemcpyAsync((void**)inputMatrix1, hostInputMatrix1, arrayBytes, hipMemcpyHostToDevice, stream[0]));
	gpuErrchk(hipMemcpyAsync((void**)inputMatrix2, hostInputMatrix2, arrayBytes, hipMemcpyHostToDevice, stream[0]));
	kernelPlusError <<< gridSize, blockSize, 0, stream[0] >>> (inputMatrix1, inputMatrix2, outputMatrix1);
	hipEventCreate(&stop[0]);
	hipEventRecord(stop[0],stream[0]);
	hipEventSynchronize(stop[0]);

	hipEventCreate(&start[1]);
  	hipEventRecord(start[1], stream[1]);

	gpuErrchk(hipMemcpyAsync((void**)redundantMatrix1, hostInputMatrix1, arrayBytesRedundant, hipMemcpyHostToDevice, stream[1]));
	gpuErrchk(hipMemcpyAsync((void**)redundantMatrix2, hostInputMatrix2, arrayBytesRedundant, hipMemcpyHostToDevice, stream[1]));
	kernel <<< redundantGridSize, blockSize, 0, stream[1] >>> (redundantMatrix1, redundantMatrix2, outputMatrix2);	

	hipEventCreate(&stop[1]);
	hipEventRecord(stop[1],stream[1]);
	hipEventSynchronize(stop[1]);

	gpuErrchk(hipMemcpyAsync(hostOutputMatrix, outputMatrix1, arrayBytes, hipMemcpyDeviceToHost, stream[0]));
	gpuErrchk(hipMemcpyAsync(hostRedundantMatrix, outputMatrix2, arrayBytesRedundant, hipMemcpyDeviceToHost, stream[1]));

	hipStreamDestroy(stream[0]);
	hipStreamDestroy(stream[1]);
	hipEventElapsedTime(&timer[0], start[0],stop[0]);
	hipEventElapsedTime(&timer[1], start[1],stop[1]);
	std::cout << "Error calculation time [ms]: " << timer[0] << std::endl;
	std::cout << "Redundant calculation time [ms]: " << timer[1] << std::endl;

	hipFree(inputMatrix1);
	hipFree(inputMatrix2);
	hipFree(redundantMatrix1);
	hipFree(redundantMatrix2);
	hipFree(outputMatrix1);
	hipFree(outputMatrix2);
}

void CellInfoToFloat(float * output, CellInfo * input, int arraySize)
{
	for (int i = 0; i < arraySize; i++)
	{
		output[i] = input[i].value;
	}
}

void testStartKernel_CellInfo()
{
	Matrix m1("matrixes/bcsstk03.mtx");
	Matrix m2("matrixes/bcsstk03.mtx");

	int arraySize = m1.getNonZeroValuesAmount();
	int arrayBytes = arraySize * sizeof(CellInfo);

	// init CPU vars // no smart pointers in .cu allowed, watch out
	CellInfo *host_mIn1 = new CellInfo[arraySize]; host_mIn1 = m1.getMatrix();
	CellInfo *host_mIn2 = new CellInfo[arraySize]; host_mIn2 = m2.getMatrix();
	CellInfo *host_mOut = new CellInfo[arraySize];

	// init GPU vars
	CellInfo *device_mIn1;
	CellInfo *device_mIn2;
	CellInfo *device_mOut;

	// alloc GPU memory
	gpuErrchk(hipMalloc((void**)&device_mIn1, arrayBytes));
	gpuErrchk(hipMalloc((void**)&device_mIn2, arrayBytes));
	gpuErrchk(hipMalloc((void**)&device_mOut, arrayBytes));
	// copy memory to device
	gpuErrchk(hipMemcpy(device_mIn1, host_mIn1, arrayBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(device_mIn2, host_mIn2, arrayBytes, hipMemcpyHostToDevice));

	// launch kernel
	kernel << <arraySize, 1 >> >(device_mIn1, device_mIn2, device_mOut);
	// launch kernel with error
	//kernelPlusError << <arraySize, 1 >> >(device_mIn1, device_mIn2, device_mOut);
	gpuErrchk(hipPeekAtLastError()); // debug

	// copy memory from device
	gpuErrchk(hipMemcpy(host_mOut, device_mOut, arrayBytes, hipMemcpyDeviceToHost));
	
	std::cout << "INPUT: \n";
	for (int i = 0; i < arraySize; i++)
		std::cout << host_mIn1[i].value << " ";
	std::cout << "\nOUTPUT: \n";
	for (int i = 0; i < arraySize; i++)
		std::cout << host_mOut[i].value<< " ";

	// cleaning
	hipFree(device_mIn1);
	hipFree(device_mIn2);
	hipFree(device_mOut);
	delete[] host_mOut;
}

void testStartKernel_float()
{
	Matrix m1("matrixes/bcsstk03.mtx");
	Matrix m2("matrixes/bcsstk03.mtx");

	int arraySize = m1.getNonZeroValuesAmount();
	int arrayBytes = arraySize * sizeof(float);

	// init CPU vars // no smart pointers in .cu allowed, watch out
	float *host_mIn1 = new float[arraySize];  CellInfoToFloat(host_mIn1, m1.getMatrix(), m1.getNonZeroValuesAmount());
	float *host_mIn2 = new float[arraySize];  CellInfoToFloat(host_mIn2, m2.getMatrix(), m2.getNonZeroValuesAmount());
	float *host_mOut = new float[arraySize];

	// init GPU vars
	float *device_mIn1;
	float *device_mIn2;
	float *device_mOut;

	// alloc GPU memory
	gpuErrchk(hipMalloc((void**)&device_mIn1, arrayBytes));
	gpuErrchk(hipMalloc((void**)&device_mIn2, arrayBytes));
	gpuErrchk(hipMalloc((void**)&device_mOut, arrayBytes));
	// copy memory to device
	gpuErrchk(hipMemcpy(device_mIn1, host_mIn1, arrayBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(device_mIn2, host_mIn2, arrayBytes, hipMemcpyHostToDevice));

	// launch kernel
	//kernel << <arraySize, 1 >> >(device_mIn1, device_mIn2, device_mOut);
	// launch kernel with error
	kernelPlusError << <arraySize, 1 >> >(device_mIn1, device_mIn2, device_mOut);
	gpuErrchk(hipPeekAtLastError()); // debug

	// copy memory from device
	gpuErrchk(hipMemcpy(host_mOut, device_mOut, arrayBytes, hipMemcpyDeviceToHost));

	std::cout << "INPUT: \n";
	for (int i = 0; i < arraySize; i++)
		std::cout << host_mIn1[i] << " ";
	std::cout << "\nOUTPUT: \n";
	for (int i = 0; i < arraySize; i++)
		std::cout << host_mOut[i] << " ";

	// cleaning
	hipFree(device_mIn1);
	hipFree(device_mIn2);
	hipFree(device_mOut);
	delete[] host_mOut;
}

int StartKernel_floatWithCounter()
{
	Matrix m1("matrixes/bcsstk03.mtx");
	Matrix m2("matrixes/bcsstk03.mtx");

	int arraySize = m1.getNonZeroValuesAmount();
	int arrayBytes = arraySize * sizeof(float);
	int tally = 0; // thread counter

	// init CPU vars // no smart pointers in .cu allowed, watch out
	float *host_mIn1 = new float[arraySize];  CellInfoToFloat(host_mIn1, m1.getMatrix(), m1.getNonZeroValuesAmount());
	float *host_mIn2 = new float[arraySize];  CellInfoToFloat(host_mIn2, m2.getMatrix(), m2.getNonZeroValuesAmount());
	float *host_mOut = new float[arraySize];

	// init GPU vars
	float *device_mIn1;
	float *device_mIn2;
	float *device_mOut;
	int *device_tally;

	// alloc GPU memory
	gpuErrchk(hipMalloc((void**)&device_mIn1, arrayBytes));
	gpuErrchk(hipMalloc((void**)&device_mIn2, arrayBytes));
	gpuErrchk(hipMalloc((void**)&device_mOut, arrayBytes));
	gpuErrchk(hipMalloc((void **)&device_tally, sizeof(int)));
	// copy memory to device
	gpuErrchk(hipMemcpy(device_mIn1, host_mIn1, arrayBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(device_mIn2, host_mIn2, arrayBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(device_tally, &tally, sizeof(int), hipMemcpyHostToDevice));

	// launch kernel
	//kernel << <arraySize, 1 >> >(device_mIn1, device_mIn2, device_mOut);
	// launch kernel with error
	kernelWithCounter << <arraySize, 1 >> >(device_mIn1, device_mIn2, device_mOut, device_tally);
	gpuErrchk(hipPeekAtLastError()); // debug

	// copy memory from device
	gpuErrchk(hipMemcpy(host_mOut, device_mOut, arrayBytes, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&tally, device_tally, sizeof(int), hipMemcpyDeviceToHost));

	// cleaning
	hipFree(device_mIn1);
	hipFree(device_mIn2);
	hipFree(device_mOut);
	hipFree(device_tally);
	delete[] host_mOut;

	return tally;
}


int StartKernel_floatWithCounter(int gridSize, int blockSize, Matrix &m1, Matrix &m2)
{
	// IMPORTANT - m1 AND m2 NEED TO HAVE EQUAL NON ZERO VALUES AMOUNT

	int arraySize = m1.getNonZeroValuesAmount();
	int arrayBytes = arraySize * sizeof(float);
	int tally = 0; // thread counter

	// init CPU vars // no smart pointers in .cu allowed, watch out
	float *host_mIn1 = new float[arraySize];  CellInfoToFloat(host_mIn1, m1.getMatrix(), m1.getNonZeroValuesAmount());
	float *host_mIn2 = new float[arraySize];  CellInfoToFloat(host_mIn2, m2.getMatrix(), m2.getNonZeroValuesAmount());
	float *host_mOut = new float[arraySize];

	// init GPU vars
	float *device_mIn1;
	float *device_mIn2;
	float *device_mOut;
	int *device_tally;

	// alloc GPU memory
	gpuErrchk(hipMalloc((void**)&device_mIn1, arrayBytes));
	gpuErrchk(hipMalloc((void**)&device_mIn2, arrayBytes));
	gpuErrchk(hipMalloc((void**)&device_mOut, arrayBytes));
	gpuErrchk(hipMalloc((void **)&device_tally, sizeof(int)));
	// copy memory to device
	gpuErrchk(hipMemcpy(device_mIn1, host_mIn1, arrayBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(device_mIn2, host_mIn2, arrayBytes, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(device_tally, &tally, sizeof(int), hipMemcpyHostToDevice));

	// launch kernel
	//kernel << <arraySize, 1 >> >(device_mIn1, device_mIn2, device_mOut);
	// launch kernel with error
	kernelWithCounter << <gridSize, blockSize >> >(device_mIn1, device_mIn2, device_mOut, device_tally);
	gpuErrchk(hipPeekAtLastError()); // debug

	// copy memory from device
	gpuErrchk(hipMemcpy(host_mOut, device_mOut, arrayBytes, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&tally, device_tally, sizeof(int), hipMemcpyDeviceToHost));

	// cleaning
	hipFree(device_mIn1);
	hipFree(device_mIn2);
	hipFree(device_mOut);
	hipFree(device_tally);
	delete[] host_mOut;

	return tally;
}